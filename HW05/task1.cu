#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__
void factorial()
{
    int fact = 1;

    int index =threadIdx.x+1;

    for (int i = 1; i < index+1; i++)
    {
        fact *= i;
    }

    std::printf("%d!=%d \n", index, fact);
}

int main()
{
    factorial<<<1, 8>>>();

    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
      return 1;
    }

    hipDeviceSynchronize();

    return 0;
}


