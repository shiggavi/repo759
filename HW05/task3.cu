#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "vscale.cuh"
#include <cstdlib>
#include <ctime>
#include <chrono>

using namespace std;

int main(int argc, char** argv)
{
    if (argc < 2) {
        cout << "Please provide the size of the array (n) as a command line argument." << endl;
        return 1;
    }

    unsigned int n = atoi(argv[1]);

    // Allocate memory for arrays on the host
    float* a = new float[n];
    float* b = new float[n];

    // Initialize random number generators
    default_random_engine gen;
    uniform_real_distribution<float> distribution1(-10.0, 10.0);
    uniform_real_distribution<float> distribution2(0.0, 1.0);

    for (unsigned int i = 0; i < n; i++) {
        a[i] = distribution1(gen);
        b[i] = distribution2(gen);
    }

    // Allocate memory on the device
    float* d_a;
    float* d_b;
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Set up the execution configuration
    dim3 blockDim(512);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

    // CUDA events for timing
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // Start timing
    hipEventRecord(start);
    
    // Launch the kernel
    vscale<<<gridDim, blockDim>>>(d_a, d_b, n);

    // End timing
    hipEventRecord(end);
    hipEventSynchronize(end);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    cout << milliseconds << " ";

    // Copy the result back to host
    hipMemcpy(b, d_b, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the first and last element of the result
    cout << b[0] << " ";
    cout << b[n - 1] << endl;

    // Free memory
    delete[] a;
    delete[] b;
    hipFree(d_a);
    hipFree(d_b);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}

