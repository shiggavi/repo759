#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <random>
#include "matmul.cuh"


int main(int argc, char *argv[])
{
    int  n = std::atoi(argv[1]);
    unsigned int threads_per_block= std::atoi(argv[2]);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);

    float *a, *b, *c;
    a=(float*)malloc(n*n*sizeof(float));
    b=(float*)malloc(n*n*sizeof(float));
    c=(float*)malloc(n*n*sizeof(float));
    for (int i=0; i<n*n; i++)
    {
        a[i] = dist(gen);
        b[i] = dist(gen);
    }

    float  *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, n * n * sizeof(float));
    hipMalloc((void**)&d_b, n * n * sizeof(float));
    hipMalloc((void**)&d_c, n * n * sizeof(float));

    hipMemcpy(d_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmul(d_a, d_b, d_c, n, threads_per_block);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float m_s = 0;
    hipEventElapsedTime(&m_s, start, stop);

    hipMemcpy(c, d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    std::cout<<c[n*n-1]<<" "<<m_s<<"\n";

    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
       fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
       return 1;
    }



    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}

