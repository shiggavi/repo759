#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <random>
#include <bits/stdc++.h>
#include "reduce.cuh"

#define BLOCK_SIZE 16

using namespace std;
using std::cout;

int main(int argc, char *argv[])
{
    unsigned int n = atoi(argv[1]);
    unsigned threads_per_block=atoi(argv[2]);
    float *A,R;
    A=(float*)malloc(n*sizeof(float));

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for(unsigned int i=0;i<n;i++)
    {
         A[i]=dist(gen);
    }

    float *input;
    hipMalloc((void**)&input, n*sizeof(float));
    hipMemcpy(input, A, n*sizeof(float), hipMemcpyHostToDevice);

    float *output;
    hipMalloc((void**)&output, sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    reduce(&input,&output,n,threads_per_block);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float m_s = 0;
    hipEventElapsedTime(&m_s, start, stop);

    hipMemcpy(&R,output, sizeof(float), hipMemcpyDeviceToHost);

    std::cout<<R<<"\n"<<m_s<<"\n";

    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
       fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
       return 1;
    }

    hipFree(input);
    hipFree(output);
    free(A);
}

