#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <random>
#include <bits/stdc++.h>
#include "matmul.cuh"


using namespace std;
using std::cout;


int main(int argc, char* argv[])
{

    unsigned int n = std::atoi(argv[1]);
    unsigned int block_dim =std::atoi(argv[2]);
    int* h_Ai = new int[n * n];
    int* h_Bi = new int[n * n];

    float* h_Af = new float[n * n];
    float* h_Bf = new float[n * n];

    double* h_Ad = new double[n * n];
    double* h_Bd = new double[n * n];

    int* h_C1 = new int[n * n];
    float* h_C2 = new float[n * n];
    double* h_C3 = new double[n * n];
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dist1(-10, 10);
    std::uniform_real_distribution<float> dist2(-10.0f, 10.0f);
    std::uniform_real_distribution<double> dist3(-10.0, 10.0);

    for (unsigned int i = 0; i < n * n; ++i)
    {
        h_Ai[i] = dist1(gen);
        h_Bi[i] = dist1(gen);
    }
    for (unsigned int i = 0; i < n * n; ++i)
    {
        h_Af[i] = dist2(gen);
        h_Bf[i] = dist2(gen);
    }

    for (unsigned int i = 0; i < n * n; ++i)
    {
        h_Ad[i] = dist3(gen);
        h_Bd[i] = dist3(gen);
    }


    int* d_Ai, *d_Bi, *d_C1;
    float* d_Af, *d_Bf, *d_C2;
    double* d_Ad, *d_Bd, *d_C3;
    hipMalloc((void**)&d_Ai, n * n * sizeof(int));
    hipMalloc((void**)&d_Bi, n * n * sizeof(int));

    hipMalloc((void**)&d_Af, n * n * sizeof(float));
    hipMalloc((void**)&d_Bf, n * n * sizeof(float));

    hipMalloc((void**)&d_Ad, n * n * sizeof(double));
    hipMalloc((void**)&d_Bd, n * n * sizeof(double));

    hipMalloc((void**)&d_C1, n * n * sizeof(int));
    hipMalloc((void**)&d_C2, n * n * sizeof(float));
    hipMalloc((void**)&d_C3, n * n * sizeof(double));

    hipMemcpy(d_Ai, h_Ai, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Bi, h_Bi, n * n * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_Af, h_Af, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Bf, h_Bf, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_Ad, h_Ad, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Bd, h_Bd, n * n * sizeof(double), hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul_1(d_Ai, d_Bi, d_C1, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds1 = 0;
    hipEventElapsedTime(&milliseconds1, start, stop);


    hipEventRecord(start);
    matmul_2(d_Af, d_Bf, d_C2, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start, stop);


    hipEventRecord(start);
    matmul_3(d_Ad, d_Bd, d_C3, n, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds3 = 0;
    hipEventElapsedTime(&milliseconds3, start, stop);


    hipMemcpy(h_C1, d_C1, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_C2, d_C2, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_C3, d_C3, n * n * sizeof(double), hipMemcpyDeviceToHost);

    std::cout << h_C1[0] << std::endl;
    std::cout << h_C1[n * n - 1] << std::endl;
    std::cout << milliseconds1 << std::endl;
    std::cout << h_C2[0] << std::endl;
    std::cout << h_C2[n * n - 1] << std::endl;
    std::cout << milliseconds2 << std::endl;
    std::cout << h_C3[0] << std::endl;
    std::cout << h_C3[n * n - 1] << std::endl;
    std::cout << milliseconds3 << std::endl;

    hipFree(d_Ai);
    hipFree(d_Bi);
    hipFree(d_Af);
    hipFree(d_Bf);
    hipFree(d_Ad);
    hipFree(d_Bd);

    hipFree(d_C1);
    hipFree(d_C2);
    hipFree(d_C3);
    delete[] h_Ai;
    delete[] h_Bi;
    delete[] h_Af;
    delete[] h_Bf;
    delete[] h_Ad;
    delete[] h_Bd;


    delete[] h_C1;
    delete[] h_C2;
    delete[] h_C3;

    return 0;
}

